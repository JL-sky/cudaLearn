#include <iostream>

#include "../utils.hpp"

constexpr int SIZE = 10 * 1024 * 1024;
/*
以下针对host侧，分别使用malloc和hipHostAlloc进行内存分配进行性能测试
*/
float CudaMallocTest(int size, bool up) {
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  float elapsed_time = 0;
  int *a;
  a = (int *)malloc(size * sizeof(*a));
  int *dev_a;
  CUDA_CHECK(hipMalloc(&dev_a, size * sizeof(*dev_a)));

  CUDA_CHECK(hipEventRecord(start));
  for (int i = 0; i < 100; ++i) {
    if (up) {
      CUDA_CHECK(
          hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    } else {
      CUDA_CHECK(
          hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }
  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
  free(a);
  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  return elapsed_time;
}

float CudaHostAllocTest(int size, bool up) {
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  float elapsed_time = 0;
  int *a;
  CUDA_CHECK(hipHostAlloc(&a, size * sizeof(*a), hipHostMallocDefault));
  int *dev_a;
  CUDA_CHECK(hipMalloc(&dev_a, size * sizeof(*dev_a)));

  CUDA_CHECK(hipEventRecord(start));
  for (int i = 0; i < 100; ++i) {
    if (up) {
      CUDA_CHECK(
          hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice));
    } else {
      CUDA_CHECK(
          hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }
  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
  CUDA_CHECK(hipHostFree(a));
  CUDA_CHECK(hipFree(dev_a));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  return elapsed_time;
}

int main() {
  float elapsed_time;
  float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;
  printf("============ host to device test =============\n");
  elapsed_time = CudaMallocTest(SIZE, true);
  printf("Time using CudaMalloc:%3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy up: %3.1f\n", MB / (elapsed_time / 1000));

  elapsed_time = CudaHostAllocTest(SIZE, true);
  printf("Time using CudaHostAlloc:%3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy up: %3.1f\n\n", MB / (elapsed_time / 1000));

  printf("============ device to host test =============\n");
  elapsed_time = CudaMallocTest(SIZE, false);
  printf("Time using CudaMalloc:%3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy up: %3.1f\n", MB / (elapsed_time / 1000));

  elapsed_time = CudaHostAllocTest(SIZE, false);
  printf("Time using CudaHostAlloc:%3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy up: %3.1f\n", MB / (elapsed_time / 1000));
  return 0;
}